#include "hip/hip_runtime.h"
﻿//opencv_cuda.cu:使用自定义函数，实现cuda版本图片翻转
//authored by ZhangXiaobo
//version：visual studio 2015\cuda toolkit 9.1\openCV 3.4.1
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include<opencv2/opencv.hpp>
#include<iostream>

using namespace std;
using namespace cv;

#ifdef _DEBUG
#pragma comment ( lib,"opencv_core341d.lib")
#pragma comment ( lib,"opencv_highgui341d.lib")
#pragma comment ( lib,"opencv_calib3d341d.lib")
#pragma comment ( lib,"opencv_imgcodecs341d.lib")
#pragma comment ( lib,"opencv_imgproc341d.lib")
#pragma comment ( lib,"opencv_cudaimgproc341d.lib")
#pragma comment ( lib,"opencv_cudaarithm341d.lib")
#pragma comment ( lib,"cudart.lib")
#else
#pragma comment ( lib,"opencv_core341.lib")
#pragma comment ( lib,"opencv_highgui341.lib")
#pragma comment ( lib,"opencv_calib3d341.lib")
#pragma comment ( lib,"opencv_imgcodecs341.lib")
#pragma comment ( lib,"opencv_imgproc341.lib")
#pragma comment ( lib,"opencv_cudaimgproc341.lib")
#pragma comment ( lib,"opencv_cudaarithm341.lib")
#pragma comment ( lib,"cudart.lib")
#endif
//出错处理函数
#define CHECK_ERROR(call){\
    const hipError_t err = call;\
    if (err != hipSuccess)\
    {\
        printf("Error:%s,%d,",__FILE__,__LINE__);\
        printf("code:%d,reason:%s\n",err,hipGetErrorString(err));\
        exit(1);\
    }\
}
//内核函数：实现上下翻转
__global__ void swap_image_kernel(cuda::PtrStepSz<uchar3> cu_src, cuda::PtrStepSz<uchar3> cu_dst, int h, int w)
{
	//计算的方法：参看前面两文
	unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;
	//为啥要这样限制：参看前面两文
	if (x < cu_src.cols && y < cu_src.rows)
	{
		//为何是h-y-1,而不是h-y，自己思考哦
		cu_dst(y, x) = cu_src(h - y - 1, x);
	}
}
//调用函数，主要处理block和grid的关系
void swap_image(cuda::GpuMat src, cuda::GpuMat dst, int h, int w)
{
	assert(src.cols == w && src.rows == h);
	int uint = 32;
	//参考前面两文的block和grid的计算方法，注意不要超过GPU限制
	dim3 block(uint, uint);
	dim3 grid((w + block.x - 1) / block.x, (h + block.y - 1) / block.y);
	printf("grid = %4d %4d %4d\n", grid.x, grid.y, grid.z);
	printf("block= %4d %4d %4d\n", block.x, block.y, block.z);
	swap_image_kernel <<< grid, block >>> (src, dst, h, w);
}
int main()
{
	Mat src, dst;
	cuda::GpuMat cu_src, cu_dst;
	int h, w;
	//根据argv[1]读入图片数据，BGR格式读进来
	src = imread("1.jpg");
	//检测是否正确读入
	if (src.data == NULL)
	{
		cout << "Read image error!\n" << endl;
		return -1;
	}
	h = src.rows; w = src.cols;
	cout << "图片高：" << h << ",图片宽：" << w << endl;
	//上传CPU图像数据到GPU，跟hipMalloc和hipMemcpy很像哦，其实upload里面就是这么写的
	cu_src.upload(src);
	//申请GPU空间，也可以到函数里申请，不管怎样总要申请，要不然内核函数会爆掉哦
	cu_dst = cuda::GpuMat(h, w, CV_8UC3, Scalar(0, 0, 0));
	//申请CPU空间
	dst = Mat(h, w, CV_8UC3, Scalar(0, 0, 0));
	//调用函数swap_image,由该函数调用内核函数，这样层次分明，不容易出错
	//当然你也可以直接在这里调用内核函数，东西太多代码容易乱
	swap_image(cu_src, cu_dst, h, w);
	//下载GPU数据到CPU，与upload()对应
	cu_dst.download(dst);
	//显示cpu图像，如果安装了openCV集成了openGL,那可以直接显示GpuMat
	imshow("dst", dst);
	//等待按键
	waitKey();
	//写图片到文件
	return 0;
}
